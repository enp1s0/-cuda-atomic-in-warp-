
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(
		std::uint32_t* const ptr,
		std::uint32_t* const src_ptr
		) {
	ptr[threadIdx.x] = atomicAdd(src_ptr, 1u);
}

void test(
		const std::uint32_t num_threads
		) {
	std::uint32_t* src_ptr;
	hipMallocManaged(&src_ptr, sizeof(std::uint32_t));
	hipMemset(src_ptr, 0, sizeof(std::uint32_t));

	std::uint32_t* result_ptr;
	hipMallocManaged(&result_ptr,   sizeof(std::uint32_t) * num_threads);
	hipMemset(result_ptr, 0, sizeof(std::uint32_t) * num_threads);

	kernel<<<1, num_threads>>>(
			result_ptr,
			src_ptr
			);

	hipDeviceSynchronize();

	for (std::uint32_t i = 0; i < num_threads; i++) {
		std::printf("[%4u] : %u\n", i, result_ptr[i]);
	}
}

int main() {
	test(128);
}
